///////////////////////////////////////////////////////////////////////////////
// proc.cu
//
// Contains definitions of functions related to proc module
///////////////////////////////////////////////////////////////////////////////

#include "proc.hpp"

#include <cstdio>

#include <hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>

void init()
{
    // Configure CUDA device, at the moment, by default using dev 0
    checkCudaErrors(hipSetDevice(0));
}

void deinit()
{
    // Release all resources acquired on the device
    checkCudaErrors(hipDeviceReset());
}

Image process_image(Image img, const ProcessConfig& config)
{
	printf("*** Processing image\n");

	return img;
}
