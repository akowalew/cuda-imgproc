///////////////////////////////////////////////////////////////////////////////
// proc.cu
//
// Contains definitions of functions related to proc module
///////////////////////////////////////////////////////////////////////////////

#include "proc.hpp"

#include <cstdio>

#include <hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>

//
// Private functions
//

static void set_device(int device)
{
	printf("*** Setting CUDA device no. %d\n", device);

    checkCudaErrors(hipSetDevice(device));
}

static void reset_device()
{
	printf("*** Resetting current CUDA device\n");

	checkCudaErrors(hipDeviceReset());
}

//
// Public functions
//

void init()
{
	printf("*** Initializing proc module\n");

	set_device(0);
}

void deinit()
{
	printf("*** Deinitializing proc module\n");

	reset_device();
}

Image process_image(Image img, const ProcessConfig& config)
{
	printf("*** Processing image\n");

	return img;
}
