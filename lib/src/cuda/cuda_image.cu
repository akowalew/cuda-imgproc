///////////////////////////////////////////////////////////////////////////////
// cuda_image.cuh
//
// Contains definitions for CUDA image manager
///////////////////////////////////////////////////////////////////////////////

#include "cuda_image.cuh"

#include <cassert>
#include <cstdio>

#include <hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>

CudaImage cuda_create_image(size_t cols, size_t rows)
{
	printf("*** Creating CUDA image of size %lux%lu\n", cols, rows);

	// Calculate parameters needed by CUDA
	const auto width = (cols * sizeof(uchar));
	const auto height = rows;

	// Perform pitched memory allocation
	void* data;
	size_t pitch;
	checkCudaErrors(hipMallocPitch(&data, &pitch, width, height));

	printf("*** Created CUDA image at 0x%p and pitch %lu\n", data, pitch);

	// Return created image
	return CudaImage { data, pitch, cols, rows };
}

void cuda_free_image(CudaImage& d_img)
{
	printf("*** Freeing CUDA image at %p\n", d_img.data);

	checkCudaErrors(hipFree(d_img.data));
}

CudaImage cuda_image_clone(const CudaImage& d_src)
{
	// Retrieve device image shape
	const auto cols = d_src.cols;
	const auto rows = d_src.rows;

	printf("*** Cloning CUDA image of size %lux%lu\n", cols, rows);

	// Allocate image on the device and copy device data
	auto d_dst = cuda_create_image(cols, rows);
	cuda_image_copy(d_dst, d_src);

	// Return cloned device image
	return d_dst;
}

CudaImage cuda_image_clone_from_host(const HostImage& h_src)
{
	// Validate host image shape
	assert(h_src.cols > 0);
	assert(h_src.cols > 0);

	// Retrieve host image shape
	const auto cols = (size_t) h_src.cols;
	const auto rows = (size_t) h_src.rows;

	printf("*** Cloning CUDA image from host of size %lux%lu\n", cols, rows);

	// Allocate image on the device and copy host data
	auto d_dst = cuda_create_image(cols, rows);
	cuda_image_copy_from_host(d_dst, h_src);

	// Return cloned host device image
	return d_dst;
}

HostImage cuda_image_clone_to_host(const CudaImage& d_src)
{
	// Retrieve device image shape
	const auto cols = d_src.cols;
	const auto rows = d_src.rows;

	printf("*** Cloning CUDA image to host of size %lux%lu\n", cols, rows);

	// Allocate image on host and copy device data
	auto h_dst = create_image(cols, rows);
	cuda_image_copy_to_host(h_dst, d_src);

	// Return cloned host image
	return h_dst;
}

void cuda_image_copy(CudaImage& d_dst, const CudaImage& d_src)
{
	// Retrieve device image shape
	const auto cols = d_dst.cols;
	const auto rows = d_dst.rows;

	printf("*** Copying CUDA image of size %lux%lu\n", cols, rows);

	// Calculate parameters needed by cuda
	const auto width = (cols * sizeof(uchar));
	const auto height = rows;

	// Perform data copy on device
	checkCudaErrors(hipMemcpy2D(d_dst.data, d_dst.pitch,
		d_src.data, d_src.pitch, width, height, hipMemcpyDeviceToDevice));
}

void cuda_image_copy_from_host(CudaImage& d_dst, const HostImage& h_src)
{
	// Ensure proper images sizes
	assert(d_dst.cols == h_src.cols);
	assert(d_dst.rows == h_src.rows);

	cuda_image_copy_data_from_host(d_dst, h_src.data);
}

void cuda_image_copy_data_from_host(CudaImage& d_dst, const void* h_src_data)
{
	const auto cols = d_dst.cols;
	const auto rows = d_dst.rows;

	printf("*** Copying CUDA image data from host of size %lux%lu\n", cols, rows);

	// Calculate parameters needed by cuda
	const auto width = (cols * sizeof(uchar));
	const auto height = rows;

	// Source's pitch is same as row width
	const auto dpitch = d_dst.pitch;
	const auto spitch = width;

	// Perform data copy from host
	checkCudaErrors(hipMemcpy2D(d_dst.data, dpitch, h_src_data, spitch,
		width, height, hipMemcpyHostToDevice));
}

void cuda_image_copy_to_host(HostImage& h_dst, const CudaImage& d_src)
{
	// Ensure proper images sizes
	assert(h_dst.cols == d_src.cols);
	assert(h_dst.rows == d_src.rows);

	cuda_image_copy_data_to_host(h_dst.data, d_src);
}

void cuda_image_copy_data_to_host(void* h_dst_data, const CudaImage& d_src)
{
	const auto cols = d_src.cols;
	const auto rows = d_src.rows;

	printf("*** Copying CUDA image data to host of size %lux%lu\n", cols, rows);

	// Calculate parameters needed by cuda
	const auto width = (cols * sizeof(uchar));
	const auto height = rows;

	// Destination's pitch is same as row width
	const auto spitch = d_src.pitch;
	const auto dpitch = width;

	// Perform data copy to host
	checkCudaErrors(hipMemcpy2D(h_dst_data, dpitch, d_src.data, spitch,
		width, height, hipMemcpyDeviceToHost));	
}

// CudaImage::CudaImage(size_t width, size_t height)
//     :   width(width)
//     ,   height(height)
// {
//     // Allocate memory for 2D image
//     checkCudaErrors(hipMallocPitch(&data, &pitch, width, height));
// }

// CudaImage::~CudaImage()
// {
//     if(data)
//     {
//         // Free memory of 2D image
//         checkCudaErrors(hipFree(data));
//     }
// }

// void CudaImage::fill(int value)
// {
//     // Fill 2D image with value
// 	checkCudaErrors(hipMemset2D(data, pitch, value, width, height));
// }

// void CudaImage::copy_from_host(const Image& img)
// {
//     // Both images must have same size
//     assert(img.cols == width);
//     assert(img.rows == height);

//     // Pitch of host image must be zero
//     assert(img.isContinuous());
//     const auto img_pitch = 0;

//     // Perform copy of image from host to device
//     checkCudaErrors(hipMemcpy2D(data, pitch,
//         img.data, img_pitch, width, height, hipMemcpyHostToDevice));
// }

// void CudaImage::copy_to_host(Image& img)
// {
//     // Both images must have same size
//     assert(img.cols == width);
//     assert(img.rows == height);

//     // Pitch of host image must be zero
//     assert(img.isContinuous());
//     const auto img_pitch = 0;

//     // Perform copy of image from device to host
//     checkCudaErrors(hipMemcpy2D(img.data, img_pitch,
//         data, pitch, width, height, hipMemcpyDeviceToHost));
// }
