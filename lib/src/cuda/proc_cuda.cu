///////////////////////////////////////////////////////////////////////////////
// proc_cuda.cu
//
// Contains definitions of functions related to CUDA image processor module
///////////////////////////////////////////////////////////////////////////////

#include "proc_cuda.cuh"

#include <cstdio>

#include <hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>

//
// Private functions
//

static void set_device(int device)
{
	printf("*** Setting CUDA device no. %d\n", device);

    checkCudaErrors(hipSetDevice(device));
}

static void reset_device()
{
	printf("*** Resetting current CUDA device\n");

	checkCudaErrors(hipDeviceReset());
}

//
// Public functions
//

void init_cuda()
{
	printf("*** Initializing proc CUDA module\n");

	set_device(0);
}

void deinit_cuda()
{
	printf("*** Deinitializing proc CUDA module\n");

	reset_device();
}

Image process_image_cuda(Image img, const ProcessConfig& config)
{
	printf("*** Processing image with CUDA\n");

	return img;
}
