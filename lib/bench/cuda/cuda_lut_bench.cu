///////////////////////////////////////////////////////////////////////////////
// cuda_lut_bench.cu
//
// Contains definitions of benchmarks for CUDA histogram equalizer
///////////////////////////////////////////////////////////////////////////////

#include <benchmark/benchmark.h>

#include <hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>

#include "cuda_proc.cuh"
#include "cuda_hist.cuh"

template<typename TFunc>
static void cuda_benchmark(benchmark::State& state, TFunc&& func)
{
    hipEvent_t start;
    hipEvent_t stop;

    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));

    float time_ms;

    for(auto _ : state)
    {
        checkCudaErrors(hipEventRecord(start));

        func();

        checkCudaErrors(hipEventRecord(stop));
        checkCudaErrors(hipEventSynchronize(stop));
        checkCudaErrors(hipEventElapsedTime(&time_ms, start, stop));
        checkCudaErrors(hipDeviceSynchronize());

        state.SetIterationTime(time_ms / 1000.0f);
    }  

    checkCudaErrors(hipEventDestroy(stop));
    checkCudaErrors(hipEventDestroy(start));
}

static void get_resolutions(benchmark::internal::Benchmark* b)
{
    b->Args({320, 240}); 
    b->Args({640, 480});
    b->Args({1024, 768});
    b->Args({1920, 1080});
    b->Args({2560, 1440});
    b->Args({3840, 2160});
}

//! Performs benchmarking of cuda_apply_lut function
static void cuda_apply_lut(benchmark::State& state)
{
    const size_t cols = state.range(0);
    const size_t rows = state.range(1);

    cuda_proc_init();
    auto src = cuda_create_image(cols, rows);
    auto dst = cuda_create_image(cols, rows);
    auto lut = cuda_create_lut();

    cuda_benchmark(state, [&dst, &src, &lut] {
        cuda_apply_lut_async(dst, src, lut);
    });

    cuda_free_lut(lut);
    cuda_free_image(dst);
    cuda_free_image(src);
    cuda_proc_deinit();
}

BENCHMARK(cuda_apply_lut)
    ->UseRealTime()
    ->UseManualTime()
    ->Apply(get_resolutions);
