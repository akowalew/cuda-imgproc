///////////////////////////////////////////////////////////////////////////////
// cuda_hist_test.cu
//
// Contains implementation of tests for CUDA hist module
///////////////////////////////////////////////////////////////////////////////

#include "doctest.h"

#include <algorithm>

#include <hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>

#include "cuda_proc.cuh"
#include "cuda_hist.cuh"

static void cuda_calculate_hist_filled_test(
    CudaHistogram::Type* h_hist_data, CudaHistogram& d_hist, CudaImage& img, 
    CudaImage::Type img_v, size_t nelems)
{
    GIVEN("Image filled with 0x00's")
    {
        cuda_image_fill_async(img, img_v);

        WHEN("Calculating a histogram")
        {
            cuda_calculate_hist_async(d_hist, img);
            checkCudaErrors(hipDeviceSynchronize());

            THEN("All counters have 0's, except one, which equals to nelems")
            {
                cuda_histogram_copy_data_to_host(h_hist_data, d_hist);
                CHECK(std::all_of(h_hist_data, h_hist_data + img_v,
                    [](uint v) { return (v == 0); }));
                CHECK(h_hist_data[img_v] == nelems);
                CHECK(std::all_of(h_hist_data + img_v + 1, h_hist_data + CudaHistogram::Size,
                    [](uint v) { return (v == 0); }));
            }
        }
    }
}

SCENARIO("Histograms may be calculated for images")
{
	const auto cols = 8;
	const auto rows = 8;
    const auto nelems = (cols * rows);

    cuda_proc_init();
    auto h_hist_data = (CudaHistogram::Type*) malloc(CudaHistogram::BufferSize);
    auto d_hist = cuda_create_histogram();
    auto img = cuda_create_image(cols, rows);

    cuda_calculate_hist_filled_test(h_hist_data, d_hist, img, 0x00, nelems);
    cuda_calculate_hist_filled_test(h_hist_data, d_hist, img, 0x01, nelems);
    cuda_calculate_hist_filled_test(h_hist_data, d_hist, img, 0xFF, nelems);

    free(h_hist_data);
    cuda_free_histogram(d_hist);
    cuda_free_image(img);
    cuda_proc_deinit();
}
