///////////////////////////////////////////////////////////////////////////////
// cuda_hist_test.cu
//
// Contains implementation of tests for CUDA hist module
///////////////////////////////////////////////////////////////////////////////

#include "doctest.h"

#include <algorithm>

#include <hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>

#include "cuda_proc.cuh"
#include "cuda_hist.cuh"

static void cuda_calculate_hist_filled_test(
    CudaHistogram::Type* h_hist_data, CudaHistogram& d_hist, CudaImage& img, 
    CudaImage::Type img_v, size_t nelems)
{
    cuda_image_fill_async(img, img_v);
    cuda_calculate_hist_async(d_hist, img);
    checkCudaErrors(hipDeviceSynchronize());

    cuda_histogram_copy_data_to_host(h_hist_data, d_hist);
    CHECK(std::all_of(h_hist_data, h_hist_data + img_v,
        [](uint v) { return (v == 0); }));
    CHECK(h_hist_data[img_v] == nelems);
    CHECK(std::all_of(h_hist_data + img_v + 1, h_hist_data + CudaHistogram::Size,
        [](uint v) { return (v == 0); }));
}

TEST_CASE("Histograms may be calculated for images")
{
	const auto cols = 8;
	const auto rows = 8;
    const auto nelems = (cols * rows);

    cuda_proc_init();
    auto h_hist_data = (CudaHistogram::Type*) malloc(CudaHistogram::BufferSize);
    auto d_hist = cuda_create_histogram();
    auto img = cuda_create_image(cols, rows);

    SUBCASE("Image filled with 0x00's has all counters 0's, except 1st, which equals to nelems")
    {
        cuda_calculate_hist_filled_test(h_hist_data, d_hist, img, 0x00, nelems);
    }
    SUBCASE("Image filled with 0x01's has all counters 0's, except 2nd, which equals to nelems")
    {
        cuda_calculate_hist_filled_test(h_hist_data, d_hist, img, 0x01, nelems);
    }
    SUBCASE("Image filled with 0x02's has all counters 0's, except 3rd, which equals to nelems")
    {
        cuda_calculate_hist_filled_test(h_hist_data, d_hist, img, 0x02, nelems);
    }
    SUBCASE("Image filled with 0xFF's has all counters 0's, except 256th, which equals to nelems")
    {
        cuda_calculate_hist_filled_test(h_hist_data, d_hist, img, 0xFF, nelems);
    }

    free(h_hist_data);
    cuda_free_histogram(d_hist);
    cuda_free_image(img);
    cuda_proc_deinit();
}

